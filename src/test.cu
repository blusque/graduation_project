#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int len, int scale, int *a, int *b)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < len * scale; i += stride)
    {
        int aIndex = i % len;
        int bIndex = i;
        a[aIndex] += b[bIndex];
    }
}

int main()
{
    constexpr int len = 100;
    constexpr int scale = 20;
    int *b, *a;
    int bHost[len * scale];
    for (int i = 0; i < len * scale; i++)
    {
        bHost[i] = 1;
    }
    int aHost[len] = {0};
    hipMalloc((void **)&a, len * sizeof(int));
    hipMalloc((void **)&b, len * scale * sizeof(int));
    hipMemcpy(a, aHost, sizeof(int) * len, hipMemcpyHostToDevice);
    hipMemcpy(b, bHost, sizeof(int) * len * scale, hipMemcpyHostToDevice);
    add<<<10, len>>>(len, scale, a, b);
    hipMemcpy(aHost, a, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(bHost, b, sizeof(int) * len * scale, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; i++)
    {
        std::cout << aHost[i] << ' ';
    }
    hipFree(a);
    hipFree(b);
    std::cout << std::endl;
    return 0;
}
